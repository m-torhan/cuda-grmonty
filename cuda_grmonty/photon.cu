/*
 * Copyright (c) 2025 Maciej Torhan <https://github.com/m-torhan>
 *
 * SPDX-License-Identifier: Apache-2.0
 */

#include "cuda_grmonty/photon.cuh"
#include "cuda_grmonty/utils.cuh"

#include "cuda_grmonty/consts.hpp"

namespace cuda_super_photon {

void alloc_photon_array(PhotonArray &photon_array, size_t n) {
    for (int i = 0; i < consts::n_dim; ++i) {
        gpuErrchk(hipMalloc((void **)&photon_array.x[i], n * sizeof(double[consts::n_dim])));
        gpuErrchk(hipMalloc((void **)&photon_array.k[i], n * sizeof(double[consts::n_dim])));
        gpuErrchk(hipMalloc((void **)&photon_array.dkdlam[i], n * sizeof(double[consts::n_dim])));
    }
    gpuErrchk(hipMalloc((void **)&photon_array.w, n * sizeof(double)));
    gpuErrchk(hipMalloc((void **)&photon_array.e, n * sizeof(double)));
    gpuErrchk(hipMalloc((void **)&photon_array.l, n * sizeof(double)));
    gpuErrchk(hipMalloc((void **)&photon_array.x1i, n * sizeof(double)));
    gpuErrchk(hipMalloc((void **)&photon_array.x2i, n * sizeof(double)));
    gpuErrchk(hipMalloc((void **)&photon_array.tau_abs, n * sizeof(double)));
    gpuErrchk(hipMalloc((void **)&photon_array.tau_scatt, n * sizeof(double)));
    gpuErrchk(hipMalloc((void **)&photon_array.n_e_0, n * sizeof(double)));
    gpuErrchk(hipMalloc((void **)&photon_array.theta_e_0, n * sizeof(double)));
    gpuErrchk(hipMalloc((void **)&photon_array.b_0, n * sizeof(double)));
    gpuErrchk(hipMalloc((void **)&photon_array.e_0, n * sizeof(double)));
    gpuErrchk(hipMalloc((void **)&photon_array.e_0_s, n * sizeof(double)));
    gpuErrchk(hipMalloc((void **)&photon_array.n_scatt, n * sizeof(int)));
}

void free_photon_array(PhotonArray &photon_array) {
    for (int i = 0; i < consts::n_dim; ++i) {
        gpuErrchk(hipFree(photon_array.x[i]));
        gpuErrchk(hipFree(photon_array.k[i]));
        gpuErrchk(hipFree(photon_array.dkdlam[i]));
    }
    gpuErrchk(hipFree(photon_array.w));
    gpuErrchk(hipFree(photon_array.e));
    gpuErrchk(hipFree(photon_array.l));
    gpuErrchk(hipFree(photon_array.x1i));
    gpuErrchk(hipFree(photon_array.x2i));
    gpuErrchk(hipFree(photon_array.tau_abs));
    gpuErrchk(hipFree(photon_array.tau_scatt));
    gpuErrchk(hipFree(photon_array.n_e_0));
    gpuErrchk(hipFree(photon_array.theta_e_0));
    gpuErrchk(hipFree(photon_array.b_0));
    gpuErrchk(hipFree(photon_array.e_0));
    gpuErrchk(hipFree(photon_array.e_0_s));
    gpuErrchk(hipFree(photon_array.n_scatt));
}

}; /* namespace cuda_super_photon */
