#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025 Maciej Torhan <https://github.com/m-torhan>
 *
 * SPDX-License-Identifier: Apache-2.0
 */

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

#include <cstdio>
#include <queue>
#include <semaphore>
#include <tuple>

#include "cuda_grmonty/consts.hpp"
#include "cuda_grmonty/harm_data.cuh"
#include "cuda_grmonty/harm_model.cuh"
#include "cuda_grmonty/photon.cuh"
#include "cuda_grmonty/proba.cuh"
#include "cuda_grmonty/radiation.cuh"
#include "cuda_grmonty/super_photon.cuh"
#include "cuda_grmonty/utils.cuh"

#include "cuda_grmonty/harm_data.hpp"
#include "cuda_grmonty/photon.hpp"
#include "cuda_grmonty/utils.hpp"

namespace cuda_super_photon {

const int n_photons = consts::cuda::n_photons;

/**
 * @brief Maximum scattering optical depth per photon (device).
 */
__device__ double dev_max_tau_scatt = 0.0;

/**
 * @brief Number of super-photons recorded (device).
 */
__device__ int dev_n_super_photon_recorded = 0;

/**
 * @brief Number of super-photons that scattered (device).
 */
__device__ int dev_n_super_photon_scatt = 0;

/**
 * @brief Device pointer to simulation header.
 */
static struct harm::Header *dev_header;

/**
 * @brief Device copy of simulation data (geometry, fluid, zones).
 */
static struct cuda_harm::Data dev_data;

/**
 * @brief Device pointer to units structure.
 */
static struct harm::Units *dev_units;

/**
 * @brief Device copy of precomputed tables for GPU calculations.
 */
static struct cuda_harm::Tables dev_tables;

/**
 * @brief Device pointer to photon spectrum accumulator.
 */
static struct harm::Spectrum *dev_spectrum;

/**
 * @enum PhotonState
 * @brief Represents the state of a photon in the simulation.
 */
enum PhotonState : uint8_t {
    Empty = 0,       /* Photon slot is empty. */
    New = 1,         /* Newly created photon, not yet initialized. */
    Initialized = 2, /* Photon initialized with position and momentum. */
    Tracked = 3      /* Photon has been propagated/tracked. */
};

/**
 * @brief Initialize CUDA random number generator states for super-photon propagation.
 *
 * Each thread initializes its own state for use in Monte Carlo sampling.
 *
 * @param rng_state Pointer to device array of random number generator states.
 */
static __global__ void init_rng(hiprandStatePhilox4_32_10_t *rng_state);

/**
 * @brief Load new photons and validate them.
 *
 * Checks if photons are in a valid state and prepares new photons for initialization.
 *
 * @param photon       Device array of existing photons.
 * @param photon_new   Device array for newly created photons.
 * @param photon_state Device array of photon states to track initialization and tracking.
 */
static __global__ void
load_validate_photon(struct PhotonArray photon, struct PhotonArray photon_new, enum PhotonState *photon_state);

/**
 * @brief Setup per-photon propagation variables before starting photon tracking.
 *
 * Computes initial local fluid properties, photon frequency, absorption, and scattering opacities, as well as bias
 * factors for weighted Monte Carlo propagation.
 *
 * @param header       Pointer to simulation header (grid, units, parameters).
 * @param data         Device copy of simulation data (geometry, fluid, zones).
 * @param units        Pointer to unit conversion structure.
 * @param tables       Precomputed tables (hotcross, k2, f) on device memory.
 * @param bias_norm    Bias normalization factor for photon weighting.
 * @param photon       Device array of photons to initialize.
 * @param photon_state Device array of photon states.
 * @param n_step       Device array for photon step counters.
 * @param fluid_n_e    Device array of local electron number densities.
 * @param theta        Device array of local electron temperatures (theta_e).
 * @param nu           Device array of photon frequencies in the fluid frame.
 * @param alpha_scatti Device array of inverse scattering opacities.
 * @param alpha_absi   Device array of inverse absorption opacities.
 * @param bi           Device array of photon bias factors.
 */
static __global__ void setup_variables(const struct harm::Header *header,
                                       const struct cuda_harm::Data data,
                                       const struct harm::Units *units,
                                       const struct cuda_harm::Tables tables,
                                       double bias_norm,
                                       struct PhotonArray photon,
                                       enum PhotonState *photon_state,
                                       int *n_step,
                                       double *fluid_n_e,
                                       double *theta,
                                       double *nu,
                                       double *alpha_scatti,
                                       double *alpha_absi,
                                       double *bi);

/**
 * @brief Apply stopping criterion to photons during propagation.
 *
 * Determines whether each photon should stop propagating based on optical depth, boundary conditions, or other
 * termination criteria.
 *
 * @param rng_state    Device array of random number generator states for stochastic checks.
 * @param header       Pointer to simulation header (grid, units, parameters).
 * @param photon       Device array of photons to check.
 * @param photon_state Device array of photon states, updated if photon should stop.
 */
static __global__ void stop_criterion(hiprandStatePhilox4_32_10_t *rng_state,
                                      const struct harm::Header *header,
                                      struct PhotonArray photon,
                                      enum PhotonState *photon_state);

/**
 * @brief Compute the propagation step size for each photon.
 *
 * Determines the distance each photon should move in this iteration based on local fluid properties, optical depths,
 * and geometry.
 *
 * @param header       Pointer to simulation header.
 * @param photon       Device array of photons to compute step sizes for.
 * @param photon_state Device array of photon states.
 * @param step_size    Device array to store computed step sizes.
 */
static __global__ void step_size(const struct harm::Header *header,
                                 struct PhotonArray photon,
                                 enum PhotonState *photon_state,
                                 double *step_size);

/**
 * @brief Advance photons along their trajectories by the computed step size.
 *
 * Updates photon positions and optionally accumulates path lengths.
 *
 * @param header       Pointer to simulation header.
 * @param photon       Device array of photons to propagate.
 * @param photon_state Device array of photon states.
 * @param dl           Device array to store path length increments for each photon.
 */
static __global__ void
push_photon(const struct harm::Header *header, struct PhotonArray photon, enum PhotonState *photon_state, double *dl);

/**
 * @brief Compute photon interactions with the fluid, including absorption and scattering increments.
 *
 * This kernel calculates optical depth contributions (scattering and absorption) for each photon in the simulation.
 * It updates local photon properties such as frequency in the fluid frame,  angle with the magnetic field, and photon
 * weights using bias factors. Photons outside the fluid or with zero interaction rates are skipped.
 *
 * @param header        Pointer to simulation header.
 * @param data          Device copy of simulation data (geometry, fluid, zones).
 * @param units         Pointer to unit conversion structure.
 * @param tables        Precomputed tables (hotcross, k2, f) on device memory.
 * @param photon        Device array of photons to interact.
 * @param photon_state  Device array of photon states.
 * @param interact_cond Device array of boolean flags indicating which photons interact.
 * @param step_size     Device array of photon step sizes.
 * @param bias_norm     Bias normalization factor.
 * @param fluid_n_e     Device array of local electron number densities.
 * @param theta         Device array of local electron temperatures.
 * @param nu            Device array of photon frequencies in the fluid frame.
 * @param alpha_scatti  Device array of inverse scattering opacities.
 * @param alpha_absi    Device array of inverse absorption opacities.
 * @param bi            Device array of photon bias factors.
 * @param d_tau_scatt   Device array of scattering optical depth increments.
 * @param d_tau_abs     Device array of absorption optical depth increments.
 * @param bias          Device array of updated photon biases after interaction.
 */
static __global__ void interact_photon(const struct harm::Header *header,
                                       const struct cuda_harm::Data data,
                                       const struct harm::Units *units,
                                       const struct cuda_harm::Tables tables,
                                       struct PhotonArray photon,
                                       enum PhotonState *photon_state,
                                       bool *interact_cond,
                                       double *step_size,
                                       double bias_norm,
                                       double *fluid_n_e,
                                       double *theta,
                                       double *nu,
                                       double *alpha_scatti,
                                       double *alpha_absi,
                                       double *bi,
                                       double *d_tau_scatt,
                                       double *d_tau_abs,
                                       double *bias);

/**
 * @brief Process photon scattering events including secondary photon generation.
 *
 * This kernel performs Monte Carlo scattering using the photon weight and optical depth increments. Photons may be
 * partially absorbed before scattering. Secondary photons are optionally created in `photon_2` for scattering events.
 * Geodesic propagation and fluid parameters are updated for post-scatter photon states.
 *
 * @param rng_state     Device RNG states for Monte Carlo sampling.
 * @param header        Simulation header.
 * @param data          Device copy of simulation data.
 * @param units         Pointer to unit conversion structure.
 * @param tables        Precomputed tables (hotcross, k2, f).
 * @param photon        Device array of photons to propagate.
 * @param photon_state  Device array of photon states.
 * @param interact_cond Flags indicating which photons should interact.
 * @param scatter_cond  Flags indicating which photons scatter.
 * @param photon_2      Device array for secondary photons generated by scattering.
 * @param photon_p      Device array for temporary photon storage.
 * @param fluid_params  Device array of fluid parameters at photon locations.
 * @param g_cov         Device array for metric connection coefficients.
 * @param step_size     Device array of photon step sizes.
 * @param bias_norm     Photon weight bias normalization factor.
 * @param theta         Device array of electron temperatures (theta_e).
 * @param nu            Device array of photon frequencies in the fluid frame.
 * @param alpha_scatti  Device array of inverse scattering opacities.
 * @param alpha_absi    Device array of inverse absorption opacities.
 * @param bi            Device array of photon bias factors.
 * @param d_tau_scatt   Device array of scattering optical depth increments.
 * @param d_tau_abs     Device array of absorption optical depth increments.
 * @param bias          Device array of updated photon biases after interaction.
 */
static __global__ void interact_photon_2(hiprandStatePhilox4_32_10_t *rng_state,
                                         const struct harm::Header *header,
                                         const struct cuda_harm::Data data,
                                         const struct harm::Units *units,
                                         const struct cuda_harm::Tables tables,
                                         struct PhotonArray photon,
                                         enum PhotonState *photon_state,
                                         bool *interact_cond,
                                         bool *scatter_cond,
                                         struct PhotonArray photon_2,
                                         struct PhotonArray photon_p,
                                         struct harm::FluidParams *fluid_params,
                                         double *g_cov,
                                         double *step_size,
                                         double bias_norm,
                                         double *theta,
                                         double *nu,
                                         double *alpha_scatti,
                                         double *alpha_absi,
                                         double *bi,
                                         double *d_tau_scatt,
                                         double *d_tau_abs,
                                         double *bias);

/**
 * @brief Scatter a photon according to local fluid properties and electron distribution.
 *
 * Updates the photon's momentum and flags after scattering.
 *
 * @param rng_state    Device RNG states.
 * @param units        Pointer to units conversion.
 * @param photon       Device array of photons to scatter.
 * @param photon_state Device array of photon states.
 * @param scatter_cond Flags indicating which photons scatter.
 * @param photon_p     Device array for temporary photon storage.
 * @param fluid_params Fluid parameters at photon locations.
 * @param g_cov        Metric connection coefficients for Lorentz transformations.
 */
static __global__ void scatter_super_photon(hiprandStatePhilox4_32_10_t *rng_state,
                                            const struct harm::Units *units,
                                            struct PhotonArray photon,
                                            enum PhotonState *photon_state,
                                            bool *scatter_cond,
                                            struct PhotonArray photon_p,
                                            struct harm::FluidParams *fluid_params,
                                            double *g_cov);

/**
 * @brief Increment photon step counters and check against max step number.
 *
 * @param n_step       Device array of photon step counters.
 * @param photon_state Device array of photon states, updated if stopping conditions met.
 */
static __global__ void incr_check_n_step(int *n_step, enum PhotonState *photon_state);

/**
 * @brief Record photons into the spectrum accumulator.
 *
 * Adds contributions from propagated photons to the device spectrum arrays.
 *
 * @param header       Simulation header.
 * @param photon       Device array of photons.
 * @param photon_state Device array of photon states.
 * @param n_step       Device array of photon step counts.
 * @param spectrum     Device array to accumulate photon spectra.
 */
static __global__ void record_super_photon(const struct harm::Header *header,
                                           struct PhotonArray photon,
                                           enum PhotonState *photon_state,
                                           int *n_step,
                                           struct harm::Spectrum *spectrum);

/**
 * @brief Device helper: advance a single photon along a step.
 *
 * @param header    Simulation header.
 * @param photon    Photon to propagate.
 * @param step_size Distance to propagate photon.
 */
static __device__ void push_photon(const struct harm::Header *header, struct photon::Photon *photon, double step_size);

/**
 * @brief Device helper: advance a single photon along a step.
 *
 * @param header    Simulation header.
 * @param photon    Photon to propagate.
 * @param step_size Distance to propagate photon.
 *
 * @return Energy and estimated errors.
 */
static __device__ std::tuple<double, double, double>
push_photon_step(const struct harm::Header *header, struct photon::Photon *photon, double step_size);

/**
 * @brief Compute photon weight bias factor for Monte Carlo propagation.
 *
 * @param bias_norm Bias normalization factor.
 * @param t_e       Local electron temperature (theta_e).
 * @param w         Photon weight.
 *
 * @return Photon bias factor.
 */
static __device__ double bias_func(double bias_norm, double t_e, double w);

/**
 * @brief Initialize photon momentum derivative dkdlam at given position.
 *
 * @param header Simulation header.
 * @param x      Photon position 4-vector.
 * @param k_con  Photon canonical momentum 4-vector.
 * @param d_k    Output derivative of momentum along geodesic.
 */
static __device__ void init_dkdlam(const struct harm::Header *header,
                                   const double (&x)[consts::n_dim],
                                   const double (&k_con)[consts::n_dim],
                                   double (&d_k)[consts::n_dim]);

/**
 * @brief Length of flattened connection coeffictients.
 */
constexpr int lconn_flat_len = 40;

/**
 * @brief Computes index of flattened connection coefficient from 3D index.
 *
 * @param i First dimension index.
 * @param j Second dimension index.
 * @param k Third dimension index.
 *
 * @returns Flattened index.
 */
__device__ __forceinline__ int lconn_flat_idx(int i, int j, int k) {
    if (j > k) {
        /* Enforce j <= k */
        int tmp = j;
        j = k;
        k = tmp;
    }
    /* Triangular number + offset */
    return 10 * i + j * (2 * consts::n_dim - j + 1) / 2 + (k - j);
}

/**
 * @brief Compute connection coefficients at a point for geodesic propagation.
 *
 * @param header Simulation header.
 * @param x      Position 4-vector.
 * @param lconn  Output 3D array of connection coefficients.
 */
static __device__ void
get_connection(const struct harm::Header *header, const double (&x)[consts::n_dim], double (&lconn)[lconn_flat_len]);

/**
 * @brief Sample a scattered photon momentum from the electron distribution.
 *
 * @param rng_state Device RNG states.
 * @param k         Incoming photon momentum 4-vector.
 * @param p         Electron momentum 4-vector.
 * @param kp        Output scattered photon momentum 4-vector.
 */
static __device__ void sample_scattered_photon(hiprandStatePhilox4_32_10_t *rng_state,
                                               const double (&k)[consts::n_dim],
                                               double (&p)[consts::n_dim],
                                               double (&kp)[consts::n_dim]);

/**
 * @brief Perform a Lorentz boost of a 4-vector from one frame to another.
 *
 * @param v  Input 4-vector in original frame.
 * @param u  Velocity 4-vector of target frame.
 * @param vp Output boosted 4-vector.
 */
static __device__ void
boost(const double (&v)[consts::n_dim], const double (&u)[consts::n_dim], double (&vp)[consts::n_dim]);

/**
 * @brief Atomic maximum for double precision numbers in device memory.
 *
 * @param addr Pointer to memory address to perform atomic max.
 * @param val  Value to compare and store if larger.
 *
 * @return Maximum value after atomic operation.
 */
static __device__ double atomic_max_double(double *addr, double val);

void alloc_memory(const struct harm::Header &header,
                  const struct harm::Data &data,
                  const struct harm::Units &units,
                  const ndarray::NDArray<double, 2> &hotcross_table,
                  const std::array<double, consts::n_e_samp + 1> &f,
                  const std::array<double, consts::n_e_samp + 1> &k2) {
    gpuErrchk(hipMalloc((void **)&dev_header, sizeof(struct harm::Header)));
    gpuErrchk(hipMalloc((void **)&dev_units, consts::cuda::threads_per_grid * sizeof(struct harm::Units)));
    gpuErrchk(hipMalloc((void **)&dev_data.k_rho, sizeof(double) * data.k_rho.size()));
    gpuErrchk(hipMalloc((void **)&dev_data.u, sizeof(double) * data.u.size()));
    gpuErrchk(hipMalloc((void **)&dev_data.u_1, sizeof(double) * data.u_1.size()));
    gpuErrchk(hipMalloc((void **)&dev_data.u_2, sizeof(double) * data.u_2.size()));
    gpuErrchk(hipMalloc((void **)&dev_data.u_3, sizeof(double) * data.u_3.size()));
    gpuErrchk(hipMalloc((void **)&dev_data.b_1, sizeof(double) * data.b_1.size()));
    gpuErrchk(hipMalloc((void **)&dev_data.b_2, sizeof(double) * data.b_2.size()));
    gpuErrchk(hipMalloc((void **)&dev_data.b_3, sizeof(double) * data.b_3.size()));
    gpuErrchk(hipMalloc((void **)&dev_tables.hotcross_table, sizeof(double) * hotcross_table.size()));
    gpuErrchk(hipMalloc((void **)&dev_tables.f, sizeof(double) * f.size()));
    gpuErrchk(hipMalloc((void **)&dev_tables.k2, sizeof(double) * k2.size()));
    gpuErrchk(hipMalloc((void **)&dev_spectrum, sizeof(struct harm::Spectrum) * consts::n_th_bins * consts::n_e_bins));

    gpuErrchk(hipMemcpy(dev_header, &header, sizeof(struct harm::Header), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_units, &units, sizeof(struct harm::Units), hipMemcpyHostToDevice));
    gpuErrchk(
        hipMemcpy(dev_data.k_rho, data.k_rho.data(), sizeof(double) * data.k_rho.size(), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_data.u, data.u.data(), sizeof(double) * data.u.size(), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_data.u_1, data.u_1.data(), sizeof(double) * data.u_1.size(), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_data.u_2, data.u_2.data(), sizeof(double) * data.u_2.size(), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_data.u_3, data.u_3.data(), sizeof(double) * data.u_3.size(), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_data.b_1, data.b_1.data(), sizeof(double) * data.b_1.size(), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_data.b_2, data.b_2.data(), sizeof(double) * data.b_2.size(), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_data.b_3, data.b_3.data(), sizeof(double) * data.b_3.size(), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_tables.hotcross_table,
                         hotcross_table.data(),
                         sizeof(double) * hotcross_table.size(),
                         hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_tables.f, f.data(), sizeof(double) * f.size(), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_tables.k2, k2.data(), sizeof(double) * k2.size(), hipMemcpyHostToDevice));
    gpuErrchk(hipMemset(dev_spectrum, 0, sizeof(harm::Spectrum) * consts::n_th_bins * consts::n_e_bins));
}

void free_memory() {
    gpuErrchk(hipFree(dev_header));
    gpuErrchk(hipFree(dev_units));
    gpuErrchk(hipFree(dev_data.k_rho));
    gpuErrchk(hipFree(dev_data.u));
    gpuErrchk(hipFree(dev_data.u_1));
    gpuErrchk(hipFree(dev_data.u_2));
    gpuErrchk(hipFree(dev_data.u_3));
    gpuErrchk(hipFree(dev_data.b_1));
    gpuErrchk(hipFree(dev_data.b_2));
    gpuErrchk(hipFree(dev_data.b_3));
    gpuErrchk(hipFree(dev_tables.hotcross_table));
    gpuErrchk(hipFree(dev_tables.f));
    gpuErrchk(hipFree(dev_tables.k2));
    gpuErrchk(hipFree(dev_spectrum));
}

void track_super_photons(double bias_norm,
                         double max_tau_scatt,
                         utils::ConcurrentQueue<photon::InitPhoton> &photon_queue,
                         std::binary_semaphore &stop_sem,
                         harm::Spectrum (&spectrum)[consts::n_th_bins][consts::n_e_bins],
                         uint64_t &n_super_photon_recorded,
                         uint64_t &n_super_photon_scatt) {
    const int grid_dim = consts::cuda::grid_dim;
    const int block_dim = consts::cuda::block_dim;

    constexpr unsigned int n_streams = 2;

    struct PhotonArray photon_new[n_streams];
    enum PhotonState *photon_state[n_streams];

    hiprandStatePhilox4_32_10_t *dev_rng_state[n_streams];

    struct PhotonArray dev_photon[n_streams];
    enum PhotonState *dev_photon_state[n_streams];
    struct PhotonArray dev_photon_new[n_streams];
    struct PhotonArray dev_photon_2[n_streams];

    int *dev_n_step[n_streams];

    double *dev_fluid_n_e[n_streams];

    double *dev_theta[n_streams];
    double *dev_nu[n_streams];
    double *dev_alpha_scatti[n_streams];
    double *dev_alpha_absi[n_streams];
    double *dev_bi[n_streams];

    double *dev_step_size[n_streams];

    bool *dev_interact_cond[n_streams];
    bool *scatter_cond[n_streams];

    bool *dev_scatter_cond[n_streams];
    double *dev_d_tau_scatt[n_streams];
    double *dev_d_tau_abs[n_streams];
    double *dev_bias[n_streams];

    struct PhotonArray photon_p[n_streams];

    struct PhotonArray dev_photon_p[n_streams];
    struct harm::FluidParams *dev_fluid_params[n_streams];
    double *dev_g_cov[n_streams];

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dev_max_tau_scatt), &max_tau_scatt, sizeof(double)));

    for (int i = 0; i < n_streams; ++i) {
        gpuErrchk(hipMalloc((void **)&dev_rng_state[i], n_photons * sizeof(hiprandStatePhilox4_32_10_t)));

        gpuErrchk(hipHostMalloc((void **)&photon_state[i], n_photons * sizeof(enum PhotonState)));

        for (int j = 0; j < consts::n_dim; ++j) {
            photon_new[i].x[j] = new double[n_photons];
            photon_new[i].k[j] = new double[n_photons];
        }
        photon_new[i].w = new double[n_photons];
        photon_new[i].e = new double[n_photons];
        photon_new[i].l = new double[n_photons];
        photon_new[i].n_e_0 = new double[n_photons];
        photon_new[i].theta_e_0 = new double[n_photons];
        photon_new[i].b_0 = new double[n_photons];
        photon_new[i].e_0 = new double[n_photons];
        photon_new[i].n_scatt = new int[n_photons];

        for (int j = 0; j < n_photons; ++j) {
            photon_state[i][j] = PhotonState::Empty;
        }

        gpuErrchk(hipHostMalloc((void **)&scatter_cond[i], n_photons * sizeof(bool)));

        for (int j = 0; j < consts::n_dim; ++j) {
            gpuErrchk(hipHostMalloc((void **)&photon_p[i].x[j], n_photons * sizeof(double)));
            gpuErrchk(hipHostMalloc((void **)&photon_p[i].k[j], n_photons * sizeof(double)));
        }
        gpuErrchk(hipHostMalloc((void **)&photon_p[i].w, n_photons * sizeof(double)));
        gpuErrchk(hipHostMalloc((void **)&photon_p[i].e, n_photons * sizeof(double)));
        gpuErrchk(hipHostMalloc((void **)&photon_p[i].l, n_photons * sizeof(double)));
        gpuErrchk(hipHostMalloc((void **)&photon_p[i].n_e_0, n_photons * sizeof(double)));
        gpuErrchk(hipHostMalloc((void **)&photon_p[i].b_0, n_photons * sizeof(double)));
        gpuErrchk(hipHostMalloc((void **)&photon_p[i].theta_e_0, n_photons * sizeof(double)));
        gpuErrchk(hipHostMalloc((void **)&photon_p[i].e_0, n_photons * sizeof(double)));
        gpuErrchk(hipHostMalloc((void **)&photon_p[i].n_scatt, n_photons * sizeof(int)));

        /* TODO: optimize memory usage by allocating only the parts that are needed */
        alloc_photon_array(dev_photon[i], n_photons);
        gpuErrchk(hipMalloc((void **)&dev_photon_state[i], n_photons * sizeof(enum PhotonState)));
        alloc_photon_array(dev_photon_new[i], n_photons);
        alloc_photon_array(dev_photon_2[i], n_photons);
        gpuErrchk(hipMalloc((void **)&dev_n_step[i], n_photons * sizeof(int)));

        gpuErrchk(hipMalloc((void **)&dev_fluid_n_e[i], n_photons * sizeof(double)));

        gpuErrchk(hipMalloc((void **)&dev_theta[i], n_photons * sizeof(double)));
        gpuErrchk(hipMalloc((void **)&dev_nu[i], n_photons * sizeof(double)));
        gpuErrchk(hipMalloc((void **)&dev_alpha_scatti[i], n_photons * sizeof(double)));
        gpuErrchk(hipMalloc((void **)&dev_alpha_absi[i], n_photons * sizeof(double)));
        gpuErrchk(hipMalloc((void **)&dev_bi[i], n_photons * sizeof(double)));

        gpuErrchk(hipMalloc((void **)&dev_step_size[i], n_photons * sizeof(double)));

        gpuErrchk(hipMalloc((void **)&dev_interact_cond[i], n_photons * sizeof(bool)));
        gpuErrchk(hipMalloc((void **)&dev_scatter_cond[i], n_photons * sizeof(bool)));
        gpuErrchk(hipMalloc((void **)&dev_d_tau_scatt[i], n_photons * sizeof(double)));
        gpuErrchk(hipMalloc((void **)&dev_d_tau_abs[i], n_photons * sizeof(double)));
        gpuErrchk(hipMalloc((void **)&dev_bias[i], n_photons * sizeof(double)));

        alloc_photon_array(dev_photon_p[i], n_photons);
        gpuErrchk(hipMalloc((void **)&dev_fluid_params[i], n_photons * sizeof(struct harm::FluidParams)));
        gpuErrchk(hipMalloc((void **)&dev_g_cov[i], n_photons * consts::n_dim * consts::n_dim * sizeof(double)));

        gpuErrchk(hipMemset(dev_photon_state[i], 0, n_photons * sizeof(enum PhotonState)));
    }

    int n_iter = 0;
    bool queue_empty = false;
    bool all_done = false;

    hipStream_t streams[n_streams];
    hipEvent_t scattered_photons_cpy_dtoh[n_streams];
    hipEvent_t scattered_photons_enq[n_streams];
    int stream_idx = 0;

    for (auto &stream : streams) {
        hipStreamCreate(&stream);
    }
    for (auto &event : scattered_photons_cpy_dtoh) {
        hipEventCreate(&event);
    }
    for (auto &event : scattered_photons_enq) {
        hipEventCreate(&event);
    }

    for (int i = 0; i < n_streams; ++i) {
        init_rng<<<grid_dim, block_dim, 0, streams[i]>>>(dev_rng_state[i]);
    }

    gpuErrchk(hipDeviceSynchronize());

    std::queue<photon::InitPhoton> buffer;

    while (true) {
        if (stop_sem.try_acquire()) {
            queue_empty = true;
        }

        /* feed photons into array */
        all_done = true;
        if (n_iter % 7 == 0) {
            photon_queue.dequeue_n(buffer, consts::cuda::n_photons - buffer.size());

            for (int i = 0; i < n_photons; ++i) {
                if (photon_state[stream_idx][i] == PhotonState::Empty && !buffer.empty()) {
                    photon::InitPhoton p = buffer.front();
                    buffer.pop();
                    for (int j = 0; j < consts::n_dim; ++j) {
                        photon_new[stream_idx].x[j][i] = p.x[j];
                        photon_new[stream_idx].k[j][i] = p.k[j];
                    }
                    photon_new[stream_idx].w[i] = p.w;
                    photon_new[stream_idx].e[i] = p.e;
                    photon_new[stream_idx].l[i] = p.l;
                    photon_new[stream_idx].n_e_0[i] = p.n_e_0;
                    photon_new[stream_idx].b_0[i] = p.b_0;
                    photon_new[stream_idx].theta_e_0[i] = p.theta_e_0;
                    photon_new[stream_idx].e_0[i] = p.e_0;
                    photon_new[stream_idx].n_scatt[i] = p.n_scatt;
                    photon_state[stream_idx][i] = PhotonState::New;
                }
                if (photon_state[stream_idx][i] != PhotonState::Empty) {
                    all_done = false;
                }
            }

            if (queue_empty && all_done) {
                break;
            }

            /* load and validate new photons */
            for (int i = 0; i < consts::n_dim; ++i) {
                gpuErrchk(hipMemcpyAsync(dev_photon_new[stream_idx].x[i],
                                          photon_new[stream_idx].x[i],
                                          n_photons * sizeof(double),
                                          hipMemcpyHostToDevice,
                                          streams[stream_idx]));
                gpuErrchk(hipMemcpyAsync(dev_photon_new[stream_idx].k[i],
                                          photon_new[stream_idx].k[i],
                                          n_photons * sizeof(double),
                                          hipMemcpyHostToDevice,
                                          streams[stream_idx]));
            }
            gpuErrchk(hipMemcpyAsync(dev_photon_new[stream_idx].w,
                                      photon_new[stream_idx].w,
                                      n_photons * sizeof(double),
                                      hipMemcpyHostToDevice,
                                      streams[stream_idx]));
            gpuErrchk(hipMemcpyAsync(dev_photon_new[stream_idx].e,
                                      photon_new[stream_idx].e,
                                      n_photons * sizeof(double),
                                      hipMemcpyHostToDevice,
                                      streams[stream_idx]));
            gpuErrchk(hipMemcpyAsync(dev_photon_new[stream_idx].l,
                                      photon_new[stream_idx].l,
                                      n_photons * sizeof(double),
                                      hipMemcpyHostToDevice,
                                      streams[stream_idx]));
            gpuErrchk(hipMemcpyAsync(dev_photon_new[stream_idx].n_e_0,
                                      photon_new[stream_idx].n_e_0,
                                      n_photons * sizeof(double),
                                      hipMemcpyHostToDevice,
                                      streams[stream_idx]));
            gpuErrchk(hipMemcpyAsync(dev_photon_new[stream_idx].b_0,
                                      photon_new[stream_idx].b_0,
                                      n_photons * sizeof(double),
                                      hipMemcpyHostToDevice,
                                      streams[stream_idx]));
            gpuErrchk(hipMemcpyAsync(dev_photon_new[stream_idx].theta_e_0,
                                      photon_new[stream_idx].theta_e_0,
                                      n_photons * sizeof(double),
                                      hipMemcpyHostToDevice,
                                      streams[stream_idx]));
            gpuErrchk(hipMemcpyAsync(dev_photon_new[stream_idx].e_0,
                                      photon_new[stream_idx].e_0,
                                      n_photons * sizeof(double),
                                      hipMemcpyHostToDevice,
                                      streams[stream_idx]));
            gpuErrchk(hipMemcpyAsync(dev_photon_new[stream_idx].n_scatt,
                                      photon_new[stream_idx].n_scatt,
                                      n_photons * sizeof(int),
                                      hipMemcpyHostToDevice,
                                      streams[stream_idx]));

            gpuErrchk(hipMemcpyAsync(dev_photon_state[stream_idx],
                                      &photon_state[stream_idx][0],
                                      n_photons * sizeof(enum PhotonState),
                                      hipMemcpyHostToDevice,
                                      streams[stream_idx]));

            load_validate_photon<<<grid_dim, block_dim, 0, streams[stream_idx]>>>(
                dev_photon[stream_idx], dev_photon_new[stream_idx], dev_photon_state[stream_idx]);

            setup_variables<<<grid_dim, block_dim, 0, streams[stream_idx]>>>(dev_header,
                                                                             dev_data,
                                                                             dev_units,
                                                                             dev_tables,
                                                                             bias_norm,
                                                                             dev_photon[stream_idx],
                                                                             dev_photon_state[stream_idx],
                                                                             dev_n_step[stream_idx],
                                                                             dev_fluid_n_e[stream_idx],
                                                                             dev_theta[stream_idx],
                                                                             dev_nu[stream_idx],
                                                                             dev_alpha_scatti[stream_idx],
                                                                             dev_alpha_absi[stream_idx],
                                                                             dev_bi[stream_idx]);
        }
        ++n_iter;

        stop_criterion<<<grid_dim, block_dim, 0, streams[stream_idx]>>>(
            dev_rng_state[stream_idx], dev_header, dev_photon[stream_idx], dev_photon_state[stream_idx]);

        for (int i = 0; i < consts::n_dim; ++i) {
            gpuErrchk(hipMemcpyAsync(dev_photon_2[stream_idx].x[i],
                                      dev_photon[stream_idx].x[i],
                                      n_photons * sizeof(double),
                                      hipMemcpyDeviceToDevice,
                                      streams[stream_idx]));
            gpuErrchk(hipMemcpyAsync(dev_photon_2[stream_idx].k[i],
                                      dev_photon[stream_idx].k[i],
                                      n_photons * sizeof(double),
                                      hipMemcpyDeviceToDevice,
                                      streams[stream_idx]));
            gpuErrchk(hipMemcpyAsync(dev_photon_2[stream_idx].dkdlam[i],
                                      dev_photon[stream_idx].dkdlam[i],
                                      n_photons * sizeof(double),
                                      hipMemcpyDeviceToDevice,
                                      streams[stream_idx]));
        }
        gpuErrchk(hipMemcpyAsync(dev_photon_2[stream_idx].e_0_s,
                                  dev_photon[stream_idx].e_0_s,
                                  n_photons * sizeof(double),
                                  hipMemcpyDeviceToDevice,
                                  streams[stream_idx]));

        step_size<<<grid_dim, block_dim, 0, streams[stream_idx]>>>(
            dev_header, dev_photon[stream_idx], dev_photon_state[stream_idx], dev_step_size[stream_idx]);

        push_photon<<<grid_dim, block_dim, 0, streams[stream_idx]>>>(
            dev_header, dev_photon[stream_idx], dev_photon_state[stream_idx], dev_step_size[stream_idx]);

        /* check stop criterion */
        stop_criterion<<<grid_dim, block_dim, 0, streams[stream_idx]>>>(
            dev_rng_state[stream_idx], dev_header, dev_photon[stream_idx], dev_photon_state[stream_idx]);

        /* allow photon to interact with matter */
        interact_photon<<<grid_dim, block_dim, 0, streams[stream_idx]>>>(dev_header,
                                                                         dev_data,
                                                                         dev_units,
                                                                         dev_tables,
                                                                         dev_photon[stream_idx],
                                                                         dev_photon_state[stream_idx],
                                                                         dev_interact_cond[stream_idx],
                                                                         dev_step_size[stream_idx],
                                                                         bias_norm,
                                                                         dev_fluid_n_e[stream_idx],
                                                                         dev_theta[stream_idx],
                                                                         dev_nu[stream_idx],
                                                                         dev_alpha_scatti[stream_idx],
                                                                         dev_alpha_absi[stream_idx],
                                                                         dev_bi[stream_idx],
                                                                         dev_d_tau_scatt[stream_idx],
                                                                         dev_d_tau_abs[stream_idx],
                                                                         dev_bias[stream_idx]);

        interact_photon_2<<<grid_dim, block_dim, 0, streams[stream_idx]>>>(dev_rng_state[stream_idx],
                                                                           dev_header,
                                                                           dev_data,
                                                                           dev_units,
                                                                           dev_tables,
                                                                           dev_photon[stream_idx],
                                                                           dev_photon_state[stream_idx],
                                                                           dev_interact_cond[stream_idx],
                                                                           dev_scatter_cond[stream_idx],
                                                                           dev_photon_2[stream_idx],
                                                                           dev_photon_p[stream_idx],
                                                                           dev_fluid_params[stream_idx],
                                                                           dev_g_cov[stream_idx],
                                                                           dev_step_size[stream_idx],
                                                                           bias_norm,
                                                                           dev_theta[stream_idx],
                                                                           dev_nu[stream_idx],
                                                                           dev_alpha_scatti[stream_idx],
                                                                           dev_alpha_absi[stream_idx],
                                                                           dev_bi[stream_idx],
                                                                           dev_d_tau_scatt[stream_idx],
                                                                           dev_d_tau_abs[stream_idx],
                                                                           dev_bias[stream_idx]);

        /* scatter */
        scatter_super_photon<<<grid_dim, block_dim, 0, streams[stream_idx]>>>(dev_rng_state[stream_idx],
                                                                              dev_units,
                                                                              dev_photon[stream_idx],
                                                                              dev_photon_state[stream_idx],
                                                                              dev_scatter_cond[stream_idx],
                                                                              dev_photon_p[stream_idx],
                                                                              dev_fluid_params[stream_idx],
                                                                              dev_g_cov[stream_idx]);

        hipEventSynchronize(scattered_photons_enq[stream_idx]);

        for (int i = 0; i < consts::n_dim; ++i) {
            gpuErrchk(hipMemcpyAsync(photon_p[stream_idx].x[i],
                                      dev_photon_p[stream_idx].x[i],
                                      n_photons * sizeof(double),
                                      hipMemcpyDeviceToHost,
                                      streams[stream_idx]));
            gpuErrchk(hipMemcpyAsync(photon_p[stream_idx].k[i],
                                      dev_photon_p[stream_idx].k[i],
                                      n_photons * sizeof(double),
                                      hipMemcpyDeviceToHost,
                                      streams[stream_idx]));
        }
        gpuErrchk(hipMemcpyAsync(photon_p[stream_idx].w,
                                  dev_photon_p[stream_idx].w,
                                  n_photons * sizeof(double),
                                  hipMemcpyDeviceToHost,
                                  streams[stream_idx]));
        gpuErrchk(hipMemcpyAsync(photon_p[stream_idx].e,
                                  dev_photon_p[stream_idx].e,
                                  n_photons * sizeof(double),
                                  hipMemcpyDeviceToHost,
                                  streams[stream_idx]));
        gpuErrchk(hipMemcpyAsync(photon_p[stream_idx].l,
                                  dev_photon_p[stream_idx].l,
                                  n_photons * sizeof(double),
                                  hipMemcpyDeviceToHost,
                                  streams[stream_idx]));
        gpuErrchk(hipMemcpyAsync(photon_p[stream_idx].n_e_0,
                                  dev_photon_p[stream_idx].n_e_0,
                                  n_photons * sizeof(double),
                                  hipMemcpyDeviceToHost,
                                  streams[stream_idx]));
        gpuErrchk(hipMemcpyAsync(photon_p[stream_idx].b_0,
                                  dev_photon_p[stream_idx].b_0,
                                  n_photons * sizeof(double),
                                  hipMemcpyDeviceToHost,
                                  streams[stream_idx]));
        gpuErrchk(hipMemcpyAsync(photon_p[stream_idx].theta_e_0,
                                  dev_photon_p[stream_idx].theta_e_0,
                                  n_photons * sizeof(double),
                                  hipMemcpyDeviceToHost,
                                  streams[stream_idx]));
        gpuErrchk(hipMemcpyAsync(photon_p[stream_idx].e_0,
                                  dev_photon_p[stream_idx].e_0,
                                  n_photons * sizeof(double),
                                  hipMemcpyDeviceToHost,
                                  streams[stream_idx]));
        gpuErrchk(hipMemcpyAsync(photon_p[stream_idx].n_scatt,
                                  dev_photon_p[stream_idx].n_scatt,
                                  n_photons * sizeof(int),
                                  hipMemcpyDeviceToHost,
                                  streams[stream_idx]));

        gpuErrchk(hipMemcpyAsync(scatter_cond[stream_idx],
                                  dev_scatter_cond[stream_idx],
                                  n_photons * sizeof(bool),
                                  hipMemcpyDeviceToHost,
                                  streams[stream_idx]));

        gpuErrchk(hipEventRecord(scattered_photons_cpy_dtoh[stream_idx], streams[stream_idx]));

        /* increment and check step num */
        incr_check_n_step<<<grid_dim, block_dim, 0, streams[stream_idx]>>>(dev_n_step[stream_idx],
                                                                           dev_photon_state[stream_idx]);

        if (n_iter % 7 == 0) {
            /* record photons */
            record_super_photon<<<grid_dim, block_dim, 0, streams[stream_idx]>>>(
                dev_header, dev_photon[stream_idx], dev_photon_state[stream_idx], dev_n_step[stream_idx], dev_spectrum);

            /* copy photon state to host */
            gpuErrchk(hipMemcpyAsync(&photon_state[stream_idx][0],
                                      dev_photon_state[stream_idx],
                                      n_photons * sizeof(enum PhotonState),
                                      hipMemcpyDeviceToHost,
                                      streams[stream_idx]));
        }

        unsigned int prev_stream_idx = (stream_idx + n_streams - 1) % n_streams;
        hipEventSynchronize(scattered_photons_cpy_dtoh[prev_stream_idx]);

        for (int i = 0; i < n_photons; ++i) {
            if (scatter_cond[prev_stream_idx][i]) {
                photon::InitPhoton p;

                for (int j = 0; j < consts::n_dim; ++j) {
                    p.x[j] = photon_p[prev_stream_idx].x[j][i];
                    p.k[j] = photon_p[prev_stream_idx].k[j][i];
                }
                p.w = photon_p[prev_stream_idx].w[i];
                p.e = photon_p[prev_stream_idx].e[i];
                p.l = photon_p[prev_stream_idx].l[i];
                p.n_e_0 = photon_p[prev_stream_idx].n_e_0[i];
                p.b_0 = photon_p[prev_stream_idx].b_0[i];
                p.theta_e_0 = photon_p[prev_stream_idx].theta_e_0[i];
                p.e_0 = photon_p[prev_stream_idx].e_0[i];
                p.n_scatt = photon_p[prev_stream_idx].n_scatt[i];

                photon_queue.force_enqueue(p);
            }
        }

        gpuErrchk(hipEventRecord(scattered_photons_enq[stream_idx], streams[stream_idx]));

        ++stream_idx;
        stream_idx %= n_streams;
    }

    for (auto &stream : streams) {
        hipStreamDestroy(stream);
    }
    for (auto &event : scattered_photons_cpy_dtoh) {
        hipEventDestroy(event);
    }
    for (auto &event : scattered_photons_enq) {
        hipEventDestroy(event);
    }

    gpuErrchk(hipMemcpy(
        spectrum, dev_spectrum, sizeof(harm::Spectrum) * consts::n_th_bins * consts::n_e_bins, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpyFromSymbol(&n_super_photon_recorded, HIP_SYMBOL(dev_n_super_photon_recorded), sizeof(int)));
    gpuErrchk(hipMemcpyFromSymbol(&n_super_photon_scatt, HIP_SYMBOL(dev_n_super_photon_scatt), sizeof(int)));

    for (int i = 0; i < n_streams; ++i) {
        gpuErrchk(hipFree(dev_rng_state[i]));

        gpuErrchk(hipHostFree(photon_state[i]));

        for (int j = 0; j < consts::n_dim; ++j) {
            delete[] photon_new[i].x[j];
            delete[] photon_new[i].k[j];
        }
        delete[] photon_new[i].w;
        delete[] photon_new[i].e;
        delete[] photon_new[i].l;
        delete[] photon_new[i].n_e_0;
        delete[] photon_new[i].b_0;
        delete[] photon_new[i].theta_e_0;

        free_photon_array(dev_photon[i]);
        gpuErrchk(hipFree(dev_photon_state[i]));
        free_photon_array(dev_photon_new[i]);
        free_photon_array(dev_photon_2[i]);
        gpuErrchk(hipFree(dev_n_step[i]));

        gpuErrchk(hipFree(dev_fluid_n_e[i]));

        gpuErrchk(hipFree(dev_theta[i]));
        gpuErrchk(hipFree(dev_nu[i]));
        gpuErrchk(hipFree(dev_alpha_scatti[i]));
        gpuErrchk(hipFree(dev_alpha_absi[i]));
        gpuErrchk(hipFree(dev_bi[i]));

        gpuErrchk(hipFree(dev_step_size[i]));

        gpuErrchk(hipFree(dev_interact_cond[i]));
        gpuErrchk(hipHostFree(scatter_cond[i]));
        gpuErrchk(hipFree(dev_scatter_cond[i]));
        gpuErrchk(hipFree(dev_d_tau_scatt[i]));
        gpuErrchk(hipFree(dev_d_tau_abs[i]));
        gpuErrchk(hipFree(dev_bias[i]));

        for (int j = 0; j < consts::n_dim; ++j) {
            gpuErrchk(hipHostFree(photon_p[i].x[j]));
            gpuErrchk(hipHostFree(photon_p[i].k[j]));
        }
        gpuErrchk(hipHostFree(photon_p[i].w));
        gpuErrchk(hipHostFree(photon_p[i].e));
        gpuErrchk(hipHostFree(photon_p[i].l));
        gpuErrchk(hipHostFree(photon_p[i].n_e_0));
        gpuErrchk(hipHostFree(photon_p[i].b_0));
        gpuErrchk(hipHostFree(photon_p[i].theta_e_0));
        gpuErrchk(hipHostFree(photon_p[i].e_0));
        gpuErrchk(hipHostFree(photon_p[i].n_scatt));

        free_photon_array(dev_photon_p[i]);
        gpuErrchk(hipFree(dev_fluid_params[i]));
        gpuErrchk(hipFree(dev_g_cov[i]));
    }
}

static __global__ void init_rng(hiprandStatePhilox4_32_10_t *rng_state) {
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    hiprand_init(123, tid, 0, &rng_state[tid]);
}

static __global__ void
load_validate_photon(struct PhotonArray photon, struct PhotonArray photon_new, enum PhotonState *photon_state) {
    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < n_photons; tid += blockDim.x * gridDim.x) {
        if (photon_state[tid] != PhotonState::New) {
            continue;
        }

#pragma unroll
        for (int i = 0; i < consts::n_dim; ++i) {
            photon.x[i][tid] = photon_new.x[i][tid];
            photon.k[i][tid] = photon_new.k[i][tid];
        }
        photon.w[tid] = photon_new.w[tid];
        photon.e[tid] = photon_new.e[tid];
        photon.e_0[tid] = photon_new.e_0[tid];
        photon.e_0_s[tid] = photon_new.e[tid];
        photon.l[tid] = photon_new.l[tid];
        photon.tau_scatt[tid] = 0.0;
        photon.tau_abs[tid] = 0.0;
        photon.x1i[tid] = photon_new.x[1][tid];
        photon.x2i[tid] = photon_new.x[2][tid];
        photon.n_e_0[tid] = photon_new.n_e_0[tid];
        photon.b_0[tid] = photon_new.b_0[tid];
        photon.theta_e_0[tid] = photon_new.theta_e_0[tid];
        photon.n_scatt[tid] = photon_new.n_scatt[tid];

        if (isnan(photon.x[0][tid]) || isnan(photon.x[1][tid]) || isnan(photon.x[2][tid]) || isnan(photon.x[3][tid]) ||
            isnan(photon.k[0][tid]) || isnan(photon.k[1][tid]) || isnan(photon.k[2][tid]) || isnan(photon.k[3][tid]) ||
            photon.w[tid] == 0.0) {
            photon_state[tid] = PhotonState::Empty;
        }
    }
}

static __global__ void setup_variables(const struct harm::Header *header,
                                       const struct cuda_harm::Data data,
                                       const struct harm::Units *units,
                                       const struct cuda_harm::Tables tables,
                                       double bias_norm,
                                       struct PhotonArray photon,
                                       enum PhotonState *photon_state,
                                       int *n_step,
                                       double *fluid_n_e,
                                       double *theta,
                                       double *nu,
                                       double *alpha_scatti,
                                       double *alpha_absi,
                                       double *bi) {
    __shared__ double g_cov[consts::cuda::block_dim][consts::n_dim][consts::n_dim];
    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < n_photons; tid += blockDim.x * gridDim.x) {
        if (photon_state[tid] != PhotonState::New) {
            continue;
        }

        const double photon_x[consts::n_dim] = {photon.x[0][tid], photon.x[1][tid], photon.x[2][tid], photon.x[3][tid]};
        const double photon_k[consts::n_dim] = {photon.k[0][tid], photon.k[1][tid], photon.k[2][tid], photon.k[3][tid]};

        cuda_harm::gcov_func(header, photon_x, g_cov[threadIdx.x]);

        harm::FluidParams fluid_params = cuda_harm::get_fluid_params(header,
                                                                     units,
                                                                     data.k_rho,
                                                                     data.u,
                                                                     data.u_1,
                                                                     data.u_2,
                                                                     data.u_3,
                                                                     data.b_1,
                                                                     data.b_2,
                                                                     data.b_3,
                                                                     photon_x,
                                                                     g_cov[threadIdx.x]);

        fluid_n_e[tid] = fluid_params.n_e;

        theta[tid] = cuda_radiation::bk_angle(
            photon_x, photon_k, fluid_params.u_cov, fluid_params.b_cov, fluid_params.b, units->b_unit);
        nu[tid] = cuda_radiation::fluid_nu(photon_x, photon_k, fluid_params.u_cov);
        alpha_scatti[tid] =
            cuda_radiation::alpha_inv_scatt(nu[tid], fluid_params.theta_e, fluid_params.n_e, tables.hotcross_table);
        alpha_absi[tid] = cuda_radiation::alpha_inv_abs(
            nu[tid], fluid_params.theta_e, fluid_params.n_e, fluid_params.b, theta[tid], tables.k2);
        bi[tid] = bias_func(bias_norm, fluid_params.theta_e, photon.w[tid]);

        double photon_dkdlam[consts::n_dim];
        init_dkdlam(header, photon_x, photon_k, photon_dkdlam);

#pragma unroll
        for (int i = 0; i < consts::n_dim; ++i) {
            photon.dkdlam[i][tid] = photon_dkdlam[i];
        }

        n_step[tid] = 0;
        photon_state[tid] = PhotonState::Initialized;
    }
}

static __global__ void stop_criterion(hiprandStatePhilox4_32_10_t *rng_state,
                                      const struct harm::Header *header,
                                      struct PhotonArray photon,
                                      enum PhotonState *photon_state) {
    double rh_ = 1.0 + sqrt(1.0 - header->a * header->a);
    double x1_min_ = log(rh_);
    double x1_max = log(consts::r_max);
    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < n_photons; tid += blockDim.x * gridDim.x) {
        if (photon_state[tid] != PhotonState::Initialized) {
            continue;
        }

        /* TODO: reduce branching */
        if (photon.x[1][tid] < x1_min_) {
            /* stop at event horizon */
            photon_state[tid] = PhotonState::Tracked;
            continue;
        }

        if (photon.x[1][tid] > x1_max) {
            /* stop at large distance */
            if (photon.w[tid] < consts::weight_min) {
                if (hiprand_uniform(&rng_state[tid]) <= 1.0 / consts::roulette) {
                    photon.w[tid] *= consts::roulette;
                } else {
                    photon.w[tid] = 0.0;
                }
            }
            photon_state[tid] = PhotonState::Tracked;
            continue;
        }

        if (photon.w[tid] < consts::weight_min) {
            if (hiprand_uniform(&rng_state[tid]) <= 1.0 / consts::roulette) {
                photon.w[tid] *= consts::roulette;
            } else {
                photon.w[tid] = 0.0;
                photon_state[tid] = PhotonState::Tracked;
                continue;
            }
        }
    }
}

static __global__ void step_size(const struct harm::Header *header,
                                 struct PhotonArray photon,
                                 enum PhotonState *photon_state,
                                 double *step_size) {
    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < n_photons; tid += blockDim.x * gridDim.x) {
        if (photon_state[tid] != PhotonState::Initialized) {
            continue;
        }

        double dl_x_1 = consts::step_eps * photon.x[1][tid] / (fabs(photon.k[1][tid]) + consts::eps);
        double dl_x_2 = consts::step_eps * fmin(photon.x[2][tid], header->x_stop[2] - photon.x[2][tid]) /
                        (fabs(photon.k[2][tid]) + consts::eps);
        double dl_x_3 = consts::step_eps / (fabs(photon.k[3][tid]) + consts::eps);

        double i_dl_x_1 = 1.0 / (fabs(dl_x_1) + consts::eps);
        double i_dl_x_2 = 1.0 / (fabs(dl_x_2) + consts::eps);
        double i_dl_x_3 = 1.0 / (fabs(dl_x_3) + consts::eps);

        step_size[tid] = 1.0 / (i_dl_x_1 + i_dl_x_2 + i_dl_x_3);
    }
}

static __global__ void push_photon(const struct harm::Header *header,
                                   struct PhotonArray photon,
                                   enum PhotonState *photon_state,
                                   double *step_size) {
    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < n_photons; tid += blockDim.x * gridDim.x) {
        if (photon_state[tid] != PhotonState::Initialized) {
            continue;
        }

        struct photon::Photon p = {
            .x = {photon.x[0][tid], photon.x[1][tid], photon.x[2][tid], photon.x[3][tid]},
            .k = {photon.k[0][tid], photon.k[1][tid], photon.k[2][tid], photon.k[3][tid]},
            .dkdlam = {photon.dkdlam[0][tid], photon.dkdlam[1][tid], photon.dkdlam[2][tid], photon.dkdlam[3][tid]},
            .e_0_s = photon.e_0_s[tid],
        };

        push_photon(header, &p, step_size[tid]);

#pragma unroll
        for (int i = 0; i < consts::n_dim; ++i) {
            photon.x[i][tid] = p.x[i];
            photon.k[i][tid] = p.k[i];
            photon.dkdlam[i][tid] = p.dkdlam[i];
        }
        photon.e_0_s[tid] = p.e_0_s;
    }
}

static __global__ void interact_photon(const struct harm::Header *header,
                                       const struct cuda_harm::Data data,
                                       const struct harm::Units *units,
                                       const struct cuda_harm::Tables tables,
                                       struct PhotonArray photon,
                                       enum PhotonState *photon_state,
                                       bool *interact_cond,
                                       double *step_size,
                                       double bias_norm,
                                       double *fluid_n_e,
                                       double *theta,
                                       double *nu,
                                       double *alpha_scatti,
                                       double *alpha_absi,
                                       double *bi,
                                       double *d_tau_scatt,
                                       double *d_tau_abs,
                                       double *bias) {
    const double hbar = consts::hpl / (2.0 * HIP_PI);
    const double d_tau_k = 2.0 * HIP_PI * units->l_unit / (consts::me * consts::cl * consts::cl / hbar);

    __shared__ double g_cov[consts::cuda::block_dim][consts::n_dim][consts::n_dim];

    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < n_photons; tid += blockDim.x * gridDim.x) {
        const double photon_x[consts::n_dim] = {photon.x[0][tid], photon.x[1][tid], photon.x[2][tid], photon.x[3][tid]};
        const double photon_k[consts::n_dim] = {photon.k[0][tid], photon.k[1][tid], photon.k[2][tid], photon.k[3][tid]};

        if (photon_state[tid] != PhotonState::Initialized) {
            continue;
        }

        interact_cond[tid] = (alpha_absi[tid] > 0.0 || alpha_scatti[tid] > 0.0 || fluid_n_e[tid] > 0.0);

        if (!interact_cond[tid]) {
            continue;
        }

        cuda_harm::gcov_func(header, photon_x, g_cov[threadIdx.x]);

        harm::FluidParams fluid_params = cuda_harm::get_fluid_params(header,
                                                                     units,
                                                                     data.k_rho,
                                                                     data.u,
                                                                     data.u_1,
                                                                     data.u_2,
                                                                     data.u_3,
                                                                     data.b_1,
                                                                     data.b_2,
                                                                     data.b_3,
                                                                     photon_x,
                                                                     g_cov[threadIdx.x]);
        bool bound_flag = fluid_params.n_e == 0.0;

        if (!bound_flag) {
            theta[tid] = cuda_radiation::bk_angle(
                photon_x, photon_k, fluid_params.u_cov, fluid_params.b_cov, fluid_params.b, units->b_unit);
            nu[tid] = cuda_radiation::fluid_nu(photon_x, photon_k, fluid_params.u_cov);
        }

        if (bound_flag || (nu[tid] < 0.0)) {
            d_tau_scatt[tid] = 0.5 * alpha_scatti[tid] * d_tau_k * step_size[tid];
            d_tau_abs[tid] = 0.5 * alpha_absi[tid] * d_tau_k * step_size[tid];
            alpha_scatti[tid] = 0.0;
            alpha_absi[tid] = 0.0;
            bias[tid] = 0.0;
            bi[tid] = 0.0;
        } else {
            double alpha_scattf =
                cuda_radiation::alpha_inv_scatt(nu[tid], fluid_params.theta_e, fluid_params.n_e, tables.hotcross_table);
            d_tau_scatt[tid] = 0.5 * (alpha_scatti[tid] + alpha_scattf) * d_tau_k * step_size[tid];
            alpha_scatti[tid] = alpha_scattf;

            double alpha_absf = cuda_radiation::alpha_inv_abs(
                nu[tid], fluid_params.theta_e, fluid_params.n_e, fluid_params.b, theta[tid], tables.k2);
            d_tau_abs[tid] = 0.5 * (alpha_absi[tid] + alpha_absf) * d_tau_k * step_size[tid];
            alpha_absi[tid] = alpha_absf;

            double bf = bias_func(bias_norm, fluid_params.theta_e, photon.w[tid]);
            bias[tid] = 0.5 * (bi[tid] + bf);
            bi[tid] = bf;
        }
    }
}

static __global__ void interact_photon_2(hiprandStatePhilox4_32_10_t *rng_state,
                                         const struct harm::Header *header,
                                         const struct cuda_harm::Data data,
                                         const struct harm::Units *units,
                                         const struct cuda_harm::Tables tables,
                                         struct PhotonArray photon,
                                         enum PhotonState *photon_state,
                                         bool *interact_cond,
                                         bool *scatter_cond,
                                         struct PhotonArray photon_2,
                                         struct PhotonArray photon_p,
                                         struct harm::FluidParams *fluid_params,
                                         double *g_cov,
                                         double *step_size,
                                         double bias_norm,
                                         double *theta,
                                         double *nu,
                                         double *alpha_scatti,
                                         double *alpha_absi,
                                         double *bi,
                                         double *d_tau_scatt,
                                         double *d_tau_abs,
                                         double *bias) {
    double g_cov_[consts::cuda::block_dim][consts::n_dim][consts::n_dim];
    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < n_photons; tid += blockDim.x * gridDim.x) {
        scatter_cond[tid] = false;

        if (photon_state[tid] != PhotonState::Initialized || !interact_cond[tid]) {
            continue;
        }

        double x1 = -log(hiprand_uniform(&rng_state[tid]));

        photon_p.w[tid] = photon.w[tid] / bias[tid];

        if (bias[tid] * d_tau_scatt[tid] > x1 && photon_p.w[tid] > consts::weight_min) {
            double frac = x1 / (bias[tid] * d_tau_scatt[tid]);

            /* apply absorption until scattering event */
            d_tau_abs[tid] *= frac;

            if (d_tau_abs[tid] > 100) {
                /* this photon has been absorbed before scattering */
                photon_state[tid] = PhotonState::Empty;
                continue;
            }

            d_tau_scatt[tid] *= frac;

            double d_tau = d_tau_abs[tid] + d_tau_scatt[tid];

            if (d_tau_abs[tid] < 1.0e-3) {
                photon.w[tid] *= (1.0 - d_tau / 24.0 * (24.0 - d_tau * (12.0 - d_tau * (4.0 - d_tau))));
            } else {
                photon.w[tid] *= exp(-d_tau);
            }

            struct photon::Photon p = {
                .x = {photon_2.x[0][tid], photon_2.x[1][tid], photon_2.x[2][tid], photon_2.x[3][tid]},
                .k = {photon_2.k[0][tid], photon_2.k[1][tid], photon_2.k[2][tid], photon_2.k[3][tid]},
                .dkdlam = {photon_2.dkdlam[0][tid],
                           photon_2.dkdlam[1][tid],
                           photon_2.dkdlam[2][tid],
                           photon_2.dkdlam[3][tid]},
                .e_0_s = photon_2.e_0_s[tid],
            };

            push_photon(header, &p, step_size[tid] * frac);

#pragma unroll
            for (int i = 0; i < consts::n_dim; ++i) {
                photon.x[i][tid] = p.x[i];
                photon.k[i][tid] = p.k[i];
                photon.dkdlam[i][tid] = p.dkdlam[i];
            }
            photon.e_0_s[tid] = p.e_0_s;

            const double photon_x[4] = {photon.x[0][tid], photon.x[1][tid], photon.x[2][tid], photon.x[3][tid]};
            const double photon_k[4] = {photon.k[0][tid], photon.k[1][tid], photon.k[2][tid], photon.k[3][tid]};

            cuda_harm::gcov_func(header, photon_x, g_cov_[threadIdx.x]);

            harm::FluidParams fluid_params_ = cuda_harm::get_fluid_params(header,
                                                                          units,
                                                                          data.k_rho,
                                                                          data.u,
                                                                          data.u_1,
                                                                          data.u_2,
                                                                          data.u_3,
                                                                          data.b_1,
                                                                          data.b_2,
                                                                          data.b_3,
                                                                          photon_x,
                                                                          g_cov_[threadIdx.x]);

            if (fluid_params_.n_e > 0.0) {
                scatter_cond[tid] = true;

                if (photon.k[0][tid] > 1.0e5 || photon.k[0][tid] < 0.0 || isnan(photon.k[0][tid]) ||
                    isnan(photon.k[1][tid]) || isnan(photon.k[3][tid])) {
                    photon.k[0][tid] = fabs(photon.k[0][tid]);
                    photon.w[tid] = 0.0;
                }

                if (photon.w[tid] < 1.0e-100) {
                    /* must have been a problem popping k back onto light cone */
                    photon_state[tid] = PhotonState::Empty;
                    continue;
                }

#pragma unroll
                for (int i = 0; i < consts::n_dim; ++i) {
#pragma unroll
                    for (int j = 0; j < consts::n_dim; ++j) {
                        g_cov[tid * consts::n_dim * consts::n_dim + i * consts::n_dim + j] = g_cov_[threadIdx.x][i][j];
                    }
                }
                fluid_params[tid] = fluid_params_;
            }

            theta[tid] = cuda_radiation::bk_angle(
                photon_x, photon_k, fluid_params_.u_cov, fluid_params_.b_cov, fluid_params_.b, units->b_unit);
            nu[tid] = cuda_radiation::fluid_nu(photon_x, photon_k, fluid_params_.u_cov);

            if (nu[tid] < 0.0) {
                alpha_scatti[tid] = 0.0;
                alpha_absi[tid] = 0.0;
            } else {
                alpha_scatti[tid] = cuda_radiation::alpha_inv_scatt(
                    nu[tid], fluid_params_.theta_e, fluid_params_.n_e, tables.hotcross_table);
                alpha_absi[tid] = cuda_radiation::alpha_inv_abs(
                    nu[tid], fluid_params_.theta_e, fluid_params_.n_e, fluid_params_.b, theta[tid], tables.k2);
            }
            bi[tid] = bias_func(bias_norm, fluid_params_.theta_e, photon.w[tid]);

        } else {
            if (d_tau_abs[tid] > 100) {
                /* this photon has been absorbed */
                photon_state[tid] = PhotonState::Empty;
                continue;
            }

            double d_tau = d_tau_abs[tid] + d_tau_scatt[tid];
            if (d_tau < 1.0e-3) {
                photon.w[tid] *= (1.0 - d_tau / 24.0 * (24.0 - d_tau * (12.0 - d_tau * (4.0 - d_tau))));
            } else {
                photon.w[tid] *= exp(-d_tau);
            }
        }

        photon.tau_abs[tid] += d_tau_abs[tid];
        photon.tau_scatt[tid] += d_tau_scatt[tid];
    }
}

static __global__ void scatter_super_photon(hiprandStatePhilox4_32_10_t *rng_state,
                                            const struct harm::Units *units,
                                            struct PhotonArray photon,
                                            enum PhotonState *photon_state,
                                            bool *scatter_cond,
                                            struct PhotonArray photon_p,
                                            struct harm::FluidParams *fluid_params,
                                            double *g_cov) {
    __shared__ double g_cov_[consts::cuda::block_dim][consts::n_dim][consts::n_dim];

    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < n_photons; tid += blockDim.x * gridDim.x) {
        if ((photon_state[tid] != PhotonState::Initialized && photon_state[tid] != PhotonState::Tracked) ||
            !scatter_cond[tid]) {
            continue;
        }

#pragma unroll
        for (int i = 0; i < consts::n_dim; ++i) {
#pragma unroll
            for (int j = 0; j < consts::n_dim; ++j) {
                g_cov_[threadIdx.x][i][j] = g_cov[tid * consts::n_dim * consts::n_dim + i * consts::n_dim + j];
            }
        }

        double b_hat_con[consts::n_dim];

        if (fluid_params[tid].b > 0.0) {
            for (int i = 0; i < consts::n_dim; ++i) {
                b_hat_con[i] = fluid_params[tid].b_con[i] / (fluid_params[tid].b / units->b_unit);
            }
        } else {
            for (int i = 0; i < consts::n_dim; ++i) {
                b_hat_con[i] = 0.0;
            }
            b_hat_con[1] = 1.0;
        }

        double e_con[consts::n_dim][consts::n_dim];
        double e_cov[consts::n_dim][consts::n_dim];

        /* local tetrad */
        cuda_tetrads::make_tetrad(fluid_params[tid].u_con, b_hat_con, g_cov_[threadIdx.x], e_con, e_cov);

        const double photon_k[4] = {photon.k[0][tid], photon.k[1][tid], photon.k[2][tid], photon.k[3][tid]};
        double k_tetrad[consts::n_dim];

        cuda_tetrads::coordinate_to_tetrad(e_cov, photon_k, k_tetrad);

        if (k_tetrad[0] > 1.0e5 || k_tetrad[0] < 0.0 || isnan(k_tetrad[1])) {
            scatter_cond[tid] = false;
            continue;
        }

        double p[consts::n_dim];
        cuda_proba::sample_electron_distr_p(&rng_state[tid], k_tetrad, p, fluid_params[tid].theta_e);

        double k_tetrad_p[consts::n_dim];
        sample_scattered_photon(&rng_state[tid], k_tetrad, p, k_tetrad_p);

        double photon_p_k[4];

        cuda_tetrads::tetrad_to_coordinate(e_con, k_tetrad_p, photon_p_k);

#pragma unroll
        for (int i = 0; i < consts::n_dim; ++i) {
            photon_p.k[i][tid] = photon_p_k[i];
        }

        if (isnan(photon_p.k[1][tid])) {
            photon_p.w[tid] = 0.0;
            scatter_cond[tid] = false;
            continue;
        }

        double tmp_k[consts::n_dim];
        k_tetrad_p[0] *= -1.0;
        cuda_tetrads::tetrad_to_coordinate(e_cov, k_tetrad_p, tmp_k);

        photon_p.e[tid] = -tmp_k[0];
        photon_p.e_0_s[tid] = -tmp_k[0];
        photon_p.l[tid] = tmp_k[3];
        photon_p.tau_abs[tid] = 0.0;
        photon_p.tau_scatt[tid] = 0.0;
        photon_p.b_0[tid] = fluid_params[tid].b;

        photon_p.x1i[tid] = photon.x[1][tid];
        photon_p.x2i[tid] = photon.x[2][tid];
        photon_p.x[0][tid] = photon.x[0][tid];
        photon_p.x[1][tid] = photon.x[1][tid];
        photon_p.x[2][tid] = photon.x[2][tid];
        photon_p.x[3][tid] = photon.x[3][tid];

        photon_p.n_e_0[tid] = photon.n_e_0[tid];
        photon_p.theta_e_0[tid] = photon.theta_e_0[tid];
        photon_p.e_0[tid] = photon.e_0[tid];
        photon_p.n_scatt[tid] = photon.n_scatt[tid] + 1;
    }
}

static __global__ void incr_check_n_step(int *n_step, enum PhotonState *photon_state) {
    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < n_photons; tid += blockDim.x * gridDim.x) {
        if (photon_state[tid] != PhotonState::Initialized) {
            continue;
        }
        ++n_step[tid];

        if (n_step[tid] > consts::max_n_step) {
            photon_state[tid] = PhotonState::Empty;
        }
    }
}

static __global__ void record_super_photon(const struct harm::Header *header,
                                           struct PhotonArray photon,
                                           enum PhotonState *photon_state,
                                           int *n_step,
                                           struct harm::Spectrum *spectrum) {
    const double x1_max = log(consts::r_max);
    const double l_e_0 = log(1.0e-12);

    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < n_photons; tid += blockDim.x * gridDim.x) {
        if (photon_state[tid] != PhotonState::Tracked) {
            continue;
        }

        photon_state[tid] = PhotonState::Empty;

        /* record criterion */
        if (photon.x[1][tid] <= x1_max || isnan(photon.w[tid]) || isnan(photon.e[tid])) {
            continue;
        }

        atomic_max_double(&dev_max_tau_scatt, photon.tau_scatt[tid]);

        double dx2 = (header->x_stop[2] - header->x_start[2]) / (2.0 * consts::n_th_bins);
        int ix2;
        if (photon.x[2][tid] < 0.5 * (header->x_start[2] + header->x_stop[2])) {
            ix2 = static_cast<int>(photon.x[2][tid] / dx2);
        } else {
            ix2 = static_cast<int>((header->x_stop[2] - photon.x[2][tid]) / dx2);
        }

        if (ix2 < 0 || ix2 >= consts::n_th_bins) {
            continue;
        }

        double l_e = log(photon.e[tid]);
        int i_e = static_cast<int>((l_e - l_e_0) / consts::spectrum::d_l_e + 2.5) - 2;

        if (i_e < 0 || i_e >= consts::n_e_bins) {
            continue;
        }

        atomicAdd(&dev_n_super_photon_recorded, 1);
        atomicAdd(&dev_n_super_photon_scatt, photon.n_scatt[tid]);

        /* sum in photon */
        const int idx = ix2 * consts::n_e_bins + i_e;

        /* TODO: optimize it using reduction */
        atomicAdd(&spectrum[idx].dn_dle, photon.w[tid]);
        atomicAdd(&spectrum[idx].de_dle, photon.w[tid] * photon.e[tid]);
        atomicAdd(&spectrum[idx].tau_abs, photon.w[tid] * photon.tau_abs[tid]);
        atomicAdd(&spectrum[idx].tau_scatt, photon.w[tid] * photon.tau_scatt[tid]);
        atomicAdd(&spectrum[idx].x1i_av, photon.w[tid] * photon.x1i[tid]);
        atomicAdd(&spectrum[idx].x2i_sq, photon.w[tid] * (photon.x2i[tid] * photon.x2i[tid]));
        atomicAdd(&spectrum[idx].x3f_sq, photon.w[tid] * (photon.x[3][tid] * photon.x[3][tid]));
        atomicAdd(&spectrum[idx].ne_0, photon.w[tid] * (photon.n_e_0[tid]));
        atomicAdd(&spectrum[idx].b_0, photon.w[tid] * (photon.b_0[tid]));
        atomicAdd(&spectrum[idx].theta_e_0, photon.w[tid] * (photon.theta_e_0[tid]));
        atomicAdd(&spectrum[idx].nscatt, photon.n_scatt[tid]);
        atomicAdd(&spectrum[idx].nph, 1.0);
    }
}

static __device__ double bias_func(double bias_norm, double t_e, double w) {
    double max = 0.5 * w / consts::weight_min;
    double avg_num_scatt = dev_n_super_photon_scatt / (1.0 * dev_n_super_photon_recorded + 1.0);
    double bias = 100.0 * t_e * t_e / (bias_norm * dev_max_tau_scatt * (avg_num_scatt + 2.0));

    if (bias < consts::tp_over_te) {
        bias = consts::tp_over_te;
    }
    if (bias > max) {
        bias = max;
    }

    return bias / consts::tp_over_te;
}

static __device__ void init_dkdlam(const struct harm::Header *header,
                                   const double (&x)[consts::n_dim],
                                   const double (&k_con)[consts::n_dim],
                                   double (&d_k)[consts::n_dim]) {
    double lconn[lconn_flat_len];

    get_connection(header, x, lconn);

#pragma unroll
    for (int i = 0; i < consts::n_dim; ++i) {
        d_k[i] =
            -2.0 * (k_con[0] * (lconn[lconn_flat_idx(i, 0, 1)] * k_con[1] + lconn[lconn_flat_idx(i, 0, 2)] * k_con[2] +
                                lconn[lconn_flat_idx(i, 0, 3)] * k_con[3]) +
                    k_con[1] * (lconn[lconn_flat_idx(i, 1, 2)] * k_con[2] + lconn[lconn_flat_idx(i, 1, 3)] * k_con[3]) +
                    lconn[lconn_flat_idx(i, 2, 3)] * k_con[2] * k_con[3]);

        d_k[i] -= (lconn[lconn_flat_idx(i, 0, 0)] * k_con[0] * k_con[0] +
                   lconn[lconn_flat_idx(i, 1, 1)] * k_con[1] * k_con[1] +
                   lconn[lconn_flat_idx(i, 2, 2)] * k_con[2] * k_con[2] +
                   lconn[lconn_flat_idx(i, 3, 3)] * k_con[3] * k_con[3]);
    }
}

static __device__ void
get_connection(const struct harm::Header *header, const double (&x)[consts::n_dim], double (&lconn)[lconn_flat_len]) {
    const double r1 = exp(x[1]);
    const double r2 = r1 * r1;
    const double r3 = r2 * r1;
    const double r4 = r3 * r1;

    double s_x;
    double c_x;
    sincos(2.0 * HIP_PI * x[2], &s_x, &c_x);

    const double th = HIP_PI * x[2] + 0.5 * (1.0 - header->h_slope) * s_x;
    const double dthdx2 = HIP_PI * (1.0 + (1.0 - header->h_slope) * c_x);
    const double d2thdx22 = -2.0 * HIP_PI * HIP_PI * (1.0 - header->h_slope) * s_x;
    const double dthdx22 = dthdx2 * dthdx2;

    double sth;
    double cth;
    sincos(th, &sth, &cth);

    const double sth2 = sth * sth;
    const double r1sth2 = r1 * sth2;
    const double sth4 = sth2 * sth2;
    const double cth2 = cth * cth;
    const double cth4 = cth2 * cth2;
    const double s2th = 2.0 * sth * cth;
    const double c2th = 2.0 * cth2 - 1.0;

    const double a = header->a;
    const double a2 = a * a;
    const double a3 = a2 * a;
    const double a4 = a3 * a;
    const double a2sth2 = a2 * sth2;
    const double a2cth2 = a2 * cth2;
    const double a4cth4 = a4 * cth4;

    const double rho2 = r2 + a2cth2;
    const double rho22 = rho2 * rho2;
    const double rho23 = rho22 * rho2;
    const double irho2 = 1.0 / rho2;
    const double irho22 = irho2 * irho2;
    const double irho23 = irho22 * irho2;
    const double irho23_dthdx2 = irho23 / dthdx2;

    const double fac1 = r2 - a2cth2;
    const double fac1_rho23 = fac1 * irho23;
    const double fac2 = a2 + 2.0 * r2 + a2 * c2th;
    const double fac3 = a2 + r1 * (-2.0 + r1);

    lconn[lconn_flat_idx(0, 0, 0)] = 2.0 * r1 * fac1_rho23;
    lconn[lconn_flat_idx(0, 0, 1)] = r1 * (2.0 * r1 + rho2) * fac1_rho23;
    lconn[lconn_flat_idx(0, 0, 2)] = -a2 * r1 * s2th * dthdx2 * irho22;
    lconn[lconn_flat_idx(0, 0, 3)] = -2.0 * a * r1sth2 * fac1_rho23;

    /* lconn[0][1][0] = lconn[0][0][1]; */
    lconn[lconn_flat_idx(0, 1, 1)] = 2.0 * r2 * (r4 + r1 * fac1 - a4cth4) * irho23;
    lconn[lconn_flat_idx(0, 1, 2)] = -a2 * r2 * s2th * dthdx2 * irho22;
    lconn[lconn_flat_idx(0, 1, 3)] = a * r1 * (-r1 * (r3 + 2.0 * fac1) + a4cth4) * sth2 * irho23;

    /* lconn[0][2][0] = lconn[0][0][2]; */
    /* lconn[0][2][1] = lconn[0][1][2]; */
    lconn[lconn_flat_idx(0, 2, 2)] = -2.0 * r2 * dthdx22 * irho2;
    lconn[lconn_flat_idx(0, 2, 3)] = a3 * r1sth2 * s2th * dthdx2 * irho22;

    /* lconn[0][3][0] = lconn[0][0][3]; */
    /* lconn[0][3][1] = lconn[0][1][3]; */
    /* lconn[0][3][2] = lconn[0][2][3]; */
    lconn[lconn_flat_idx(0, 3, 3)] = 2.0 * r1sth2 * (-r1 * rho22 + a2sth2 * fac1) * irho23;

    lconn[lconn_flat_idx(1, 0, 0)] = fac3 * fac1 / (r1 * rho23);
    lconn[lconn_flat_idx(1, 0, 1)] = fac1 * (-2.0 * r1 + a2sth2) * irho23;
    lconn[lconn_flat_idx(1, 0, 2)] = 0.0;
    lconn[lconn_flat_idx(1, 0, 3)] = -a * sth2 * fac3 * fac1 / (r1 * rho23);

    /* lconn[1][1][0] = lconn[1][0][1]; */
    lconn[lconn_flat_idx(1, 1, 1)] =
        (r4 * (-2.0 + r1) * (1.0 + r1) + a2 * (a2 * r1 * (1.0 + 3.0 * r1) * cth4 + a4cth4 * cth2 + r3 * sth2 +
                                               r1 * cth2 * (2.0 * r1 + 3.0 * r3 - a2sth2))) *
        irho23;
    lconn[lconn_flat_idx(1, 1, 2)] = -a2 * dthdx2 * s2th / fac2;
    lconn[lconn_flat_idx(1, 1, 3)] =
        a * sth2 * (a4 * r1 * cth4 + r2 * (2.0 * r1 + r3 - a2sth2) + a2cth2 * (2.0 * r1 * (-1.0 + r2) + a2sth2)) *
        irho23;

    /* lconn[1][2][0] = lconn[1][0][2]; */
    /* lconn[1][2][1] = lconn[1][1][2]; */
    lconn[lconn_flat_idx(1, 2, 2)] = -fac3 * dthdx22 * irho2;
    lconn[lconn_flat_idx(1, 2, 3)] = 0.0;

    /* lconn[1][3][0] = lconn[1][0][3]; */
    /* lconn[1][3][1] = lconn[1][1][3]; */
    /* lconn[1][3][2] = lconn[1][2][3]; */
    lconn[lconn_flat_idx(1, 3, 3)] = -fac3 * sth2 * (r1 * rho22 - a2 * fac1 * sth2) / (r1 * rho23);

    lconn[lconn_flat_idx(2, 0, 0)] = -a2 * r1 * s2th * irho23_dthdx2;
    lconn[lconn_flat_idx(2, 0, 1)] = r1 * lconn[lconn_flat_idx(2, 0, 0)];
    lconn[lconn_flat_idx(2, 0, 2)] = 0.0;
    lconn[lconn_flat_idx(2, 0, 3)] = a * r1 * (a2 + r2) * s2th * irho23_dthdx2;

    /* lconn[2][1][0] = lconn[2][0][1]; */
    lconn[lconn_flat_idx(2, 1, 1)] = r2 * lconn[lconn_flat_idx(2, 0, 0)];
    lconn[lconn_flat_idx(2, 1, 2)] = r2 * irho2;
    lconn[lconn_flat_idx(2, 1, 3)] =
        (a * r1 * cth * sth * (r3 * (2.0 + r1) + a2 * (2.0 * r1 * (1.0 + r1) * cth2 + a2 * cth4 + 2.0 * r1sth2))) *
        irho23_dthdx2;

    /* lconn[2][2][0] = lconn[2][0][2]; */
    /* lconn[2][2][1] = lconn[2][1][2]; */
    lconn[lconn_flat_idx(2, 2, 2)] = -a2 * cth * sth * dthdx2 * irho2 + d2thdx22 / dthdx2;
    lconn[lconn_flat_idx(2, 2, 3)] = 0.0;

    /* lconn[2][3][0] = lconn[2][0][3]; */
    /* lconn[2][3][1] = lconn[2][1][3]; */
    /* lconn[2][3][2] = lconn[2][2][3]; */
    lconn[lconn_flat_idx(2, 3, 3)] =
        -cth * sth * (rho23 + a2sth2 * rho2 * (r1 * (4.0 + r1) + a2cth2) + 2.0 * r1 * a4 * sth4) * irho23_dthdx2;

    lconn[lconn_flat_idx(3, 0, 0)] = a * fac1_rho23;
    lconn[lconn_flat_idx(3, 0, 1)] = r1 * lconn[lconn_flat_idx(3, 0, 0)];
    lconn[lconn_flat_idx(3, 0, 2)] = -2.0 * a * r1 * cth * dthdx2 / (sth * rho22);
    lconn[lconn_flat_idx(3, 0, 3)] = -a2sth2 * fac1_rho23;

    /* lconn[3][1][0] = lconn[3][0][1]; */
    lconn[lconn_flat_idx(3, 1, 1)] = a * r2 * fac1_rho23;
    lconn[lconn_flat_idx(3, 1, 2)] =
        -2 * a * r1 * (a2 + 2.0 * r1 * (2.0 + r1) + a2 * c2th) * cth * dthdx2 / (sth * fac2 * fac2);
    lconn[lconn_flat_idx(3, 1, 3)] = r1 * (r1 * rho22 - a2sth2 * fac1) * irho23;

    /* lconn[3][2][0] = lconn[3][0][2]; */
    /* lconn[3][2][1] = lconn[3][1][2]; */
    lconn[lconn_flat_idx(3, 2, 2)] = -a * r1 * dthdx22 * irho2;
    lconn[lconn_flat_idx(3, 2, 3)] = dthdx2 * (0.25 * fac2 * fac2 * cth / sth + a2 * r1 * s2th) * irho22;

    /* lconn[3][3][0] = lconn[3][0][3]; */
    /* lconn[3][3][1] = lconn[3][1][3]; */
    /* lconn[3][3][2] = lconn[3][2][3]; */
    lconn[lconn_flat_idx(3, 3, 3)] = (-a * r1sth2 * rho22 + a3 * sth4 * fac1) * irho23;
}

static __device__ void push_photon(const struct harm::Header *header, struct photon::Photon *photon, double dl) {
    if (photon->x[1] < header->x_start[1]) {
        return;
    }

    double dl_stack[8] = {dl};
    int depth_stack[8] = {0};
    int n = 0;

    double x_cpy[consts::n_dim];
    double k_cpy[consts::n_dim];
    double dk_cpy[consts::n_dim];

    while (n >= 0) {
#pragma unroll
        for (int i = 0; i < consts::n_dim; ++i) {
            x_cpy[i] = photon->x[i];
            k_cpy[i] = photon->k[i];
            dk_cpy[i] = photon->dkdlam[i];
        }

        auto [e_1, err, err_e] = push_photon_step(header, photon, dl_stack[n]);

        if (depth_stack[n] < 7 && (err_e > 1.0e-4 || err > consts::e_tol || !isfinite(err))) {
#pragma unroll
            for (int i = 0; i < consts::n_dim; ++i) {
                photon->x[i] = x_cpy[i];
                photon->k[i] = k_cpy[i];
                photon->dkdlam[i] = dk_cpy[i];
            }
            dl_stack[n] = dl_stack[n] / 2.0;
            dl_stack[n + 1] = dl_stack[n];
            depth_stack[n] = depth_stack[n] + 1;
            depth_stack[n + 1] = depth_stack[n];
        } else {
            photon->e_0_s = e_1;
            --n;
        }
    }
}

static __device__ std::tuple<double, double, double>
push_photon_step(const struct harm::Header *header, struct photon::Photon *photon, double dl) {
    const double dl_2 = 0.5 * dl;
    double k[consts::n_dim];

#pragma unroll
    for (int i = 0; i < consts::n_dim; ++i) {
        double dk = photon->dkdlam[i] * dl_2;
        photon->k[i] += dk;
        k[i] = photon->k[i] + dk;
        photon->x[i] += photon->k[i] * dl;
    }

    double lconn[lconn_flat_len];

    get_connection(header, photon->x, lconn);

    double err;
    int iter = 0;

    do {
        ++iter;

        err = 0.0;

#pragma unroll
        for (int i = 0; i < consts::n_dim; ++i) {
            photon->dkdlam[i] =
                -2.0 * (k[0] * (lconn[lconn_flat_idx(i, 0, 1)] * k[1] + lconn[lconn_flat_idx(i, 0, 2)] * k[2] +
                                lconn[lconn_flat_idx(i, 0, 3)] * k[3]) +
                        k[1] * (lconn[lconn_flat_idx(i, 1, 2)] * k[2] + lconn[lconn_flat_idx(i, 1, 3)] * k[3]) +
                        lconn[lconn_flat_idx(i, 2, 3)] * k[2] * k[3]);
            photon->dkdlam[i] -=
                (lconn[lconn_flat_idx(i, 0, 0)] * k[0] * k[0] + lconn[lconn_flat_idx(i, 1, 1)] * k[1] * k[1] +
                 lconn[lconn_flat_idx(i, 2, 2)] * k[2] * k[2] + lconn[lconn_flat_idx(i, 3, 3)] * k[3] * k[3]);

            double old_k = k[i];
            k[i] = fma(dl_2, photon->dkdlam[i], photon->k[i]);
            err += fabs((old_k - k[i]) / (k[i] + consts::eps));
        }
    } while (err > consts::e_tol && iter < consts::max_iter);

#pragma unroll
    for (int i = 0; i < consts::n_dim; ++i) {
        photon->k[i] = k[i];
    }

    double g_cov_0[consts::n_dim];

    cuda_harm::gcov_0_func(header, photon->x, g_cov_0);

    /* clang-format off */
    double e_1 = -(
        photon->k[0] * g_cov_0[0]
      + photon->k[1] * g_cov_0[1]
      + photon->k[2] * g_cov_0[2]
      + photon->k[3] * g_cov_0[3]);
    /* clang-format on */

    double err_e = fabs((e_1 - photon->e_0_s) / photon->e_0_s);

    return {e_1, err, err_e};
}

static __device__ void sample_scattered_photon(hiprandStatePhilox4_32_10_t *rng_state,
                                               const double (&k)[consts::n_dim],
                                               double (&p)[consts::n_dim],
                                               double (&kp)[consts::n_dim]) {
    double ke[consts::n_dim];

    boost(k, p, ke);

    double k0p;
    double c_th;

    if (ke[0] > 1.0e-4) {
        k0p = cuda_proba::sample_klein_nishina(rng_state, ke[0]);
        c_th = 1.0 - 1.0 / k0p + 1.0 / ke[0];
    } else {
        k0p = ke[0];
        c_th = cuda_proba::sample_thomson(rng_state);
    }
    double s_th = sqrt(abs(1.0 - c_th * c_th));

    double v0x = ke[1] / ke[0];
    double v0y = ke[2] / ke[0];
    double v0z = ke[3] / ke[0];

    double n0x;
    double n0y;
    double n0z;
    cuda_proba::sample_rand_dir(rng_state, &n0x, &n0y, &n0z);

    double n0dotv0 = v0x * n0x + v0y * n0y + v0z * n0z;

    /* unit vector 2 */
    double v1x = n0x - (n0dotv0)*v0x;
    double v1y = n0y - (n0dotv0)*v0y;
    double v1z = n0z - (n0dotv0)*v0z;
    double v1 = sqrt(v1x * v1x + v1y * v1y + v1z * v1z);
    v1x /= v1;
    v1y /= v1;
    v1z /= v1;

    /* find one more unit vector using cross product;
       this guy is automatically normalized */
    double v2x = v0y * v1z - v0z * v1y;
    double v2y = v0z * v1x - v0x * v1z;
    double v2z = v0x * v1y - v0y * v1x;

    /* now resolve new momentum vector along unit vectors */
    /* create a four-vector $p$ */
    /* solve for orientation of scattered photon */

    /* find phi for new photon */
    double phi = 2.0 * HIP_PI * hiprand_uniform(rng_state);
    double s_phi = sin(phi);
    double c_phi = cos(phi);

    p[1] *= -1.;
    p[2] *= -1.;
    p[3] *= -1.;

    double dir1 = c_th * v0x + s_th * (c_phi * v1x + s_phi * v2x);
    double dir2 = c_th * v0y + s_th * (c_phi * v1y + s_phi * v2y);
    double dir3 = c_th * v0z + s_th * (c_phi * v1z + s_phi * v2z);

    double kpe[consts::n_dim] = {
        k0p,
        k0p * dir1,
        k0p * dir2,
        k0p * dir3,
    };

    /* transform k back to lab frame */
    boost(kpe, p, kp);
}

static __device__ void
boost(const double (&v)[consts::n_dim], const double (&u)[consts::n_dim], double (&vp)[consts::n_dim]) {
    double g = u[0];
    double v_ = sqrt(abs(1.0 - 1.0 / (g * g)));
    double n1 = u[1] / (g * v_ + consts::eps);
    double n2 = u[2] / (g * v_ + consts::eps);
    double n3 = u[3] / (g * v_ + consts::eps);
    double gm1 = g - 1.0;

    /* general Lorentz boost into frame u from lab frame */
    vp[0] = u[0] * v[0] - u[1] * v[1] - u[2] * v[2] - u[3] * v[3];
    vp[1] = -u[1] * v[0] + (1.0 + n1 * n1 * gm1) * v[1] + n1 * n2 * gm1 * v[2] + n1 * n3 * gm1 * v[3];
    vp[2] = -u[2] * v[0] + n2 * n1 * gm1 * v[1] + (1.0 + n2 * n2 * gm1) * v[2] + n2 * n3 * gm1 * v[3];
    vp[3] = -u[3] * v[0] + n3 * n1 * gm1 * v[1] + n3 * n2 * gm1 * v[2] + (1.0 + n3 * n3 * gm1) * v[3];
}

static __device__ double atomic_max_double(double *addr, double val) {
    /* NOLINTBEGIN */
    unsigned long long int *addr_as_ull = (unsigned long long int *)addr;

    unsigned long long int old = *addr_as_ull;
    unsigned long long int assumed;
    /* NOLINTEND */

    do {
        assumed = old;
        double old_val = __longlong_as_double(assumed);
        if (old_val >= val) {
            break; // already bigger
        }
        old = atomicCAS(addr_as_ull, assumed, __double_as_longlong(val));
    } while (assumed != old);

    return __longlong_as_double(old);
}

}; /* namespace cuda_super_photon */
